#include "hip/hip_runtime.h"
#include "kernel_none_optimize.h"
#define TILE_WIDTH 32




// __global__ void unroll_kernel(int channel_in, int height_in, int width_in, int height_kernel, 
//                             int width_kernel, int height_out, int width_out, 
//                             float* X, float* X_unroll)
// {
//     // int t = blockIdx.x * blockDim.x + threadIdx.x; //1
//     // int width_unroll = height_out * width_out; //2*2 
//     // if(t < channel_in*width_unroll)
//     // {
//     //     int c = t / width_unroll; //0 
//     //     int col_unroll = t % width_unroll;//1
//     //     int row_out = col_unroll / width_out;//0
//     //     int col_out = col_unroll % width_out;//1
//     //     int a0 = c*(width_in*height_in);//0
//     //     int w_base = c * width_kernel * height_kernel; //0
//     //     for (int p = 0; p < height_kernel; p++){ 
//     //         int a1 = ( row_out + p)*width_in; // 0
//     //         for(int q = 0; q < width_kernel; q++){
//     //             int a2 =  col_out + q; //1
//     //             int row_unroll = w_base + p * width_kernel + q; 
//     //             X_unroll[row_unroll*width_unroll + col_unroll] = X[a0 + a1 + a2];
//     //         }
//     //     }
//     // }
//     int t = blockIdx.x * blockDim.x + threadIdx.x; //
//     int height_unroll = height_out * width_out; //2
//     if(t < channel_in*height_unroll)
//     {   
//         //output is a vector size : imagearea x (kernalarea * channel_in)

//         //which chanel are we using?
//         int c = t / height_unroll; //
//         //Which row are we using?
//         int row_unroll = t % height_unroll;//

//         //start position 
//         int row_out = row_unroll / width_out;//
//         int col_out = row_unroll % width_out;//

//         //channel start position 
//         int a0 = c*(width_in*height_in);//

//         //how many rows of the channel before this?
//         int w_base =  c * width_kernel * height_kernel; //
//         for (int p = 0; p < height_kernel; p++){ 
//             int a1 = ( row_out + p)*width_in; // 
//             for(int q = 0; q < width_kernel; q++){
//                 int a2 =  col_out + q; //
//                 int col_unroll =w_base + p * width_kernel + q; //+

//                 //Attention, in spite of each channel (vector) store data in row-major, 
//                 //But our output is a matrix, so we need to perform storing in col-major
//                 //I hate this =.= 
//                 X_unroll[col_unroll*height_unroll + row_unroll] = X[a0 + a1 + a2];
//             }
//         }
//     }
// }
// __global__ void matrix_multiplication_kernel2(float* A, float* B, float* C, int m, int n, int k)
// {
// 	int c = blockIdx.x * blockDim.x + threadIdx.x; 
// 	int r = blockIdx.y * blockDim.y + threadIdx.y; 
// 	if (r < m && c < k) {
//     float sum = 0 ;
//     for (int i = 0; i < n ; i++) { 
//       sum += A[i*m + r] * B[c*n + i];
//     }
//     C[c*m + r] = sum ; 

//   } 
//   	// __shared__ float s_A[TILE_WIDTH][TILE_WIDTH];
// 	// __shared__ float s_B[TILE_WIDTH][TILE_WIDTH];
// 	// int c = blockIdx.x * blockDim.x + threadIdx.x; 
// 	// int r = blockIdx.y * blockDim.y + threadIdx.y; 
// 	// float sum = 0 ;
// 	// for(int b = 0 ; b < (n-1)/TILE_WIDTH + 1 ; b++){
//     //     if (r<m && b*TILE_WIDTH+threadIdx.x<n)
//     //         s_A[threadIdx.y][threadIdx.x] = A[r + (b*TILE_WIDTH+threadIdx.x)*m];
//     //     else
//     //         s_A[threadIdx.y][threadIdx.x] = 0;
//     //     if (b*TILE_WIDTH+threadIdx.y<n && c < k)
//     //         s_B[threadIdx.y][threadIdx.x] = B[(b*TILE_WIDTH + threadIdx.y) + c*n];
//     //     else
//     //         s_B[threadIdx.y][threadIdx.x] = 0;
//     //     __syncthreads();
        
// 	// 	if (r<m && c<k){
//     //         for(int j = 0; j < TILE_WIDTH; ++j)
//     //             sum += s_A[threadIdx.y][j] * s_B[j][threadIdx.x];
//     //     }
        
//     //         __syncthreads();
// 	// }
//     // if (r<m && c<k)
//     //     C[c * m + r] = sum; 
// }

// __host__ void Kernel_none_optimize::none_optimize_unroll(int channel_in, int height_in, int width_in, int height_kernel, 
//                             int width_kernel, int height_out, int width_out, 
//                             float* X, float* X_unroll)
// {
//     // Allocate device memory
//     //this->printDeviceInfo();
//     float *device_input, *device_output;
//     CHECK(hipMalloc((void **)&device_input, channel_in * height_in * width_in * sizeof(float)));
//     CHECK(hipMalloc((void **)&device_output, height_out * width_out * height_kernel * width_kernel * channel_in * sizeof(float)));

//     // Copy input and mask data to device
//     CHECK(hipMemcpy(device_input, X, channel_in * height_in * width_in * sizeof(float), hipMemcpyHostToDevice));

//     // Set the kernel dimensions and call the kernel
//     dim3 num_threads_per_block(1024);
//     dim3 num_blocks_in_grid((height_out * width_out  * channel_in-1)/1024 + 1 );

//     // Launch the kernel
//     unroll_kernel<<<num_blocks_in_grid, num_threads_per_block>>>( channel_in,  height_in,  width_in,  height_kernel, 
//                              width_kernel,  height_out,  width_out, 
//                             device_input,  device_output);
//     CHECK(hipDeviceSynchronize()); // Ensure that the GPU has completed the computation

//     // Copy the output back to host
//     CHECK(hipMemcpy(X_unroll, device_output, height_out * width_out * height_kernel * width_kernel * channel_in * sizeof(float), hipMemcpyDeviceToHost));

//     // Free device memory
//     CHECK(hipFree(device_input));
//     CHECK(hipFree(device_output));
// }


// __host__ void Kernel_none_optimize::none_optimize_matrix_multiplication(float* A, float* B, float* C, int m, int n, int k,
//                          dim3 blockSize )
// {
//     // Allocate device memory
//     //this->printDeviceInfo();
//     float* d_A, * d_B, * d_C;
//     CHECK(hipMalloc(&d_A, m * n * sizeof(float)));
//     CHECK(hipMalloc(&d_B, n * k * sizeof(float)));
//     CHECK(hipMalloc(&d_C, m * k * sizeof(float)));

//     // TODO: Copy data to device memories
//     CHECK(hipMemcpy(d_A, A, m * n * sizeof(float), hipMemcpyHostToDevice));
//     CHECK(hipMemcpy(d_B, B, n * k * sizeof(float), hipMemcpyHostToDevice));
    
//     dim3 gridSize((k-1)/blockSize.y + 1,(m-1)/blockSize.x + 1,1); // TODO: Compute gridSize
    
//         matrix_multiplication_kernel2<<<gridSize, blockSize>>>(d_A, d_B, d_C, m, n, k);

//     // TODO: Copy result from device memory
//     CHECK(hipMemcpy(C, d_C, m * k * sizeof(float), hipMemcpyDeviceToHost));

//     // TODO: Free device memories
//     hipFree(d_A);
//     hipFree(d_B);
//     hipFree(d_C);
// }



__global__ void conv_forward_kernel(int channel_in, int height_in, int width_in, int height_kernel, 
                            int width_kernel, int height_out,int width_out,int channel_out,
                            const float *input_data, float*unroll_matrix, const float *weight_data, float *output_data) //float* X, float* X_unroll 
{
    int t = blockIdx.x * blockDim.x + threadIdx.x; //
    int height_unroll = height_out * width_out; //2
    int width_unroll = height_kernel * width_kernel * channel_in;
    if(t < channel_in*height_unroll)
    {   
        //output is a vector size : imagearea x (kernalarea * channel_in)

        //which chanel are we using?
        int c = t / height_unroll; //
        //Which row are we using?
        int row_unroll = t % height_unroll;//

        //start position 
        int row_out = row_unroll / width_out;//
        int col_out = row_unroll % width_out;//

        //channel start position 
        int a0 = c*(width_in*height_in);//

        //how many rows of the channel before this?
        int w_base =  c * width_kernel * height_kernel; //

        for (int p = 0; p < height_kernel; p++){ 
            int a1 = ( row_out + p)*width_in; // 
            for(int q = 0; q < width_kernel; q++){
                int a2 =  col_out + q; //
                int col_unroll =w_base + p * width_kernel + q; //+

                //Attention, in spite of each channel (vector) store data in row-major, 
                //But our output is a matrix, so we need to perform storing in col-major
                //I hate this =.= 
                //X_unroll[col_unroll*height_unroll + row_unroll] = X[a0 + a1 + a2];
                unroll_matrix[col_unroll*height_unroll + row_unroll] = input_data[a0 + a1 + a2];
            }
        }
    }
    __syncthreads();
    if (t < height_unroll * channel_out) {
        int c = t % channel_out ; 
        int r = t / channel_out;
        //if (r < m && c < k) {
            float sum = 0 ;
            for (int i = 0; i < width_unroll ; i++) { 
                sum += unroll_matrix[i*height_unroll + r] * weight_data[c*width_unroll + i];
            }
            output_data[c*height_unroll + r] = sum ; 
        //}
    }
}

__host__ void Kernel_none_optimize::conv_forward_gpu_full( int n_samples,  int channel_in,  int height_in, int width_in,
                                    int height_kernel, int width_kernel,  int channel_out,
                                     float *input_data, float *weight_data, float *output_data){

    const int height_out = height_in - height_kernel + 1;
    const int width_out = width_in - width_kernel + 1;

    // Allocate device memory
    float *device_input, *device_output, *device_weight, *device_unroll_matrix;
    CHECK(hipMalloc((void **)&device_input, n_samples * channel_in * height_in * width_in * sizeof(float)));
    CHECK(hipMalloc((void **)&device_output, n_samples * channel_out * height_out * width_out * sizeof(float)));
    CHECK(hipMalloc((void **)&device_weight, channel_out * channel_in * height_kernel * width_kernel * sizeof(float)));
    CHECK(hipMalloc((void **)&device_unroll_matrix, height_out * width_out * channel_in * height_kernel * width_kernel * sizeof(float)));

    // Copy input and mask data to device
    CHECK(hipMemcpy(device_input, input_data, n_samples * channel_in * height_in * width_in * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(device_weight, weight_data, channel_out * channel_in * height_kernel * width_kernel * sizeof(float), hipMemcpyHostToDevice));

    // // Set the kernel dimensions and call the kernel
    // int height_grid = (height_out + TILE_WIDTH - 1) / TILE_WIDTH;
    // int width_grid = (width_out + TILE_WIDTH - 1) / TILE_WIDTH;
    // int Z = height_grid * width_grid;
    // dim3 num_threads_per_block(TILE_WIDTH, TILE_WIDTH, 1);
    // dim3 num_blocks_in_grid(n_samples, output_channel, Z);
    
    dim3 num_threads_per_block(1024);
    dim3 num_blocks_in_grid((height_out * width_out  * max(channel_in,channel_out)-1)/1024 + 1 );

    for (int i = 0; i < n_samples; i ++) {
        conv_forward_kernel<<<num_blocks_in_grid, num_threads_per_block>>>(channel_in, height_in, width_in, height_kernel, 
                            width_kernel, height_out, width_out, channel_out,
                            device_input + i*channel_in * height_in * width_in, device_unroll_matrix, 
                            device_weight + i*channel_in * height_in * width_in, device_output + i*channel_in * height_in * width_in);
    }

    // Launch the kernel
    //conv_forward_kernel<<<num_blocks_in_grid, num_threads_per_block>>>(device_output, device_input, device_weight, n_samples, output_channel, channel_in, height_in, width_in, kernel_height);
    CHECK(hipDeviceSynchronize()); // Ensure that the GPU has completed the computation

    // Copy the output back to host
    CHECK(hipMemcpy(output_data, device_output, n_samples * channel_out * height_out * width_out * sizeof(float), hipMemcpyDeviceToHost));

    // Free device memory
    CHECK(hipFree(device_input));
    CHECK(hipFree(device_output));
    CHECK(hipFree(device_weight));
    CHECK(hipFree(device_unroll_matrix));

}