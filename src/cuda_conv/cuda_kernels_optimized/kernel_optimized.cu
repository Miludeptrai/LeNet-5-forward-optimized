#include "hip/hip_runtime.h"
#include "kernel_optimized.h"
#define TILE_WIDTH 32


__global__ void unroll_kernel_4(int channel_in, int height_in, int width_in, int height_kernel, 
                            int width_kernel, int height_out, int width_out, 
                            float* __restrict__ input_data, float* __restrict__ unroll_matrix)
{
    int batch_idx = blockIdx.z;

    int t = blockIdx.x * blockDim.x + threadIdx.x; //
    int height_unroll = height_out * width_out; //2
    int hw_kernel = width_kernel * height_kernel;
    int width_unroll = hw_kernel * channel_in;
    int hw_in = height_in * width_in;
    if(t <channel_in* hw_in)
    {   
        //output is a vector size : imagearea x (kernalarea * channel_in)

        //which chanel are we using?
        int c = t / hw_in; //
        //ith of each channel?
        int ith = t % hw_in;//

        //start position 
        int row_in = ith / width_in;//
        int col_in = ith % width_in;//

        float in_value = input_data[batch_idx*channel_in*height_in*width_in + t];

        int row_u,col_u,p,q;
        for (p=0;p<height_kernel;p++){
            for (q=0;q<width_kernel;q++){
                //Our goal is putting value of `in_value` in correct positions
                //best performance
                //also great algorithms for parallel 
                row_u =  row_in -p;
                col_u = col_in - q;
                if(row_u < height_out && col_u < width_out && row_u >= 0 && col_u >= 0)
                    unroll_matrix[batch_idx*width_unroll*height_unroll + (c * hw_kernel + p*width_kernel + q)*height_unroll+ row_u *width_out + col_u] = in_value;
            }
        }
    }
}


__global__ void multi_weight_add_bias_kernel_2(float* unroll_matrix, float *weight_data, float* output_data,float* bias_data,
                                                int height_unroll, int width_unroll,int channel_out)//m,n,k
{
    __shared__ float s_A[TILE_WIDTH][TILE_WIDTH];
	__shared__ float s_B[TILE_WIDTH][TILE_WIDTH];
	int c = blockIdx.x * blockDim.x + threadIdx.x; 
	int r = blockIdx.y * blockDim.y + threadIdx.y; 

    int batch_idx = blockIdx.z;
    int start_unroll = batch_idx * height_unroll * width_unroll;
    int start_output = batch_idx * height_unroll * channel_out;

	float sum = 0 ;
	for(int b = 0 ; b < (width_unroll-1)/TILE_WIDTH + 1 ; b++){
        if (r<height_unroll && b*TILE_WIDTH+threadIdx.x<width_unroll)
            s_A[threadIdx.y][threadIdx.x] = unroll_matrix[start_unroll + r + (b*TILE_WIDTH+threadIdx.x)*height_unroll];
        else
            s_A[threadIdx.y][threadIdx.x] = 0;
        if (b*TILE_WIDTH+threadIdx.y<width_unroll && c < channel_out)
            s_B[threadIdx.y][threadIdx.x] = weight_data[(b*TILE_WIDTH + threadIdx.y) + c*width_unroll];
        else
            s_B[threadIdx.y][threadIdx.x] = 0;
        __syncthreads();
        
        for(int j = 0; j < TILE_WIDTH; ++j)
            sum += s_A[threadIdx.y][j] * s_B[j][threadIdx.x];
        __syncthreads();
	}
    if (r<height_unroll && c<channel_out)
        output_data[start_output + c * height_unroll + r] = sum + bias_data[c]; 
}





__host__ void Kernel_optimized::cuda_conv_forward( int n_samples,  int channel_in,  int height_in, int width_in,    
                                    int height_kernel, int width_kernel,  int channel_out,
                                     float *input_data, float *weight_data,float *bias_data, float *output_data){

    const int height_out = height_in - height_kernel + 1;
    const int width_out = width_in - width_kernel + 1;

    // Allocate device memory
    float *device_weight,*device_bias, *device_unroll_matrix; 
    CHECK(hipMalloc((void **)&device_weight, channel_out * channel_in * height_kernel * width_kernel * sizeof(float)));
    CHECK(hipMalloc((void **)&device_bias, channel_out * sizeof(float)));

    // Copy input and mask data to device
    CHECK(hipMemcpy(device_weight, weight_data, channel_out * channel_in * height_kernel * width_kernel * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(device_bias, bias_data, channel_out * sizeof(float), hipMemcpyHostToDevice));



    //what is this? yes we have n_samples images, but in some case, we cannot load them all into GPU mem,
    //we seperate them to each batch, an here I set it 32 
    int batch_size = 32;
    // setting cuda streams
    int nStreams = 4;
    float **device_input = new float*[nStreams], **device_output = new float*[nStreams];
    hipStream_t streams[nStreams];
    
    // Set the kernel dimensions and call the kernel

    // If u use another unroll_kernel, remember to change `gridSize_unroll`

    // dim3 blockSize_unroll(1024);
    // dim3 gridSize_unroll((height_out * width_out  * channel_in-1)/1024 + 1 ,1,batch_size);
    dim3 blockSize_unroll(1024);
    dim3 gridSize_unroll((height_in * width_in  * channel_in-1)/1024 + 1 ,1,batch_size);

    dim3 blockSize_multi(32, 32);
    dim3 gridSize_multi(( channel_out-1)/blockSize_multi.y + 1,(height_out * width_out-1)/blockSize_multi.x + 1,batch_size);


    for (int i = 0; i < nStreams; i++){
		CHECK(hipStreamCreate(&streams[i]));    
        //Each stream use its GPU mem, and no new GPU location
        CHECK(hipMalloc((void **)&device_input[i], batch_size * channel_in * height_in * width_in * sizeof(float)));
        CHECK(hipMalloc((void **)&device_output[i], batch_size * channel_out * height_out * width_out * sizeof(float)));
    }
    
    CHECK(hipMalloc((void **)&device_unroll_matrix, batch_size * height_out * width_out * channel_in * height_kernel * width_kernel * sizeof(float)));


    // loop through each sample
    for (int stream = 0; stream < nStreams; stream++){
        for (int i = stream * batch_size; i < n_samples; i+=nStreams*batch_size) {
            //There is a problem. Most of time, the final batch dont have enough image, will it cause error?
            //The answer is no, because there are still some images from batch before the last
            int start_in = i * channel_in * height_in * width_in;
            int start_out = i * channel_out * height_out * width_out;
            
            //copy the data to correct stream mem 
            CHECK(hipMemcpyAsync(device_input[stream], input_data + start_in, min(batch_size,n_samples-i) * channel_in * height_in * width_in * sizeof(float), hipMemcpyHostToDevice, streams[stream]));
            unroll_kernel_4<<<gridSize_unroll, blockSize_unroll, 0, streams[stream]>>>
                            (channel_in,  height_in,  width_in,  height_kernel, 
                             width_kernel,  height_out,  width_out, 
                            device_input[stream],  device_unroll_matrix);
            multi_weight_add_bias_kernel_2<<<gridSize_multi,blockSize_multi, 0, streams[stream]>>>
                                (device_unroll_matrix,device_weight,device_output[stream],device_bias
                                ,height_out * width_out, height_kernel * width_kernel * channel_in, channel_out);
                    
            CHECK(hipMemcpyAsync(output_data + start_out, device_output[stream], min(batch_size,n_samples-i) * channel_out * height_out * width_out * sizeof(float), hipMemcpyDeviceToHost, streams[stream]));
        }
    }
    hipError_t errSync = hipGetLastError();
    hipError_t errAsync = hipDeviceSynchronize();
    if (errSync != hipSuccess) 
        printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
    if (errAsync != hipSuccess)
        printf("Async kernel error: %s\n", hipGetErrorString(errAsync));
    // Free device memory
    for (int i = 0; i < nStreams; i++){
        CHECK(hipStreamSynchronize(streams[i]));
        hipStreamDestroy(streams[i]);
        //delete each stream GPU mem 
        CHECK(hipFree(device_input[i]));
        CHECK(hipFree(device_output[i]));
    }

    CHECK(hipFree(device_bias));

    CHECK(hipFree(device_weight));
    CHECK(hipFree(device_unroll_matrix));

}