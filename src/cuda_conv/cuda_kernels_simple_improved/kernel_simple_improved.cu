#include "hip/hip_runtime.h"
#include "kernel_simple_improved.h"
#define TILE_WIDTH 32

#define MAX_CONSTANT_SIZE 8192 

__constant__ float dc_weight[MAX_CONSTANT_SIZE];

__global__ void conv_forward_kernel_2(int channel_in,int height_in, int width_in, int height_kernel, 
                            int width_kernel, int height_out, int width_out, int channel_out,
                            float *input_data,  float *weight_data,float *bias_data, float *output_data)
{
    //int batch_idx = blockIdx.z;
    int out_channel_ith = blockIdx.y;
    int width_grid = (width_out - 1) / TILE_WIDTH + 1 ;
    int width_tiled = width_kernel+ TILE_WIDTH -1;

    //remember that gridSize.x is used to perform height_out*witdh_out pixel, but in one demension
    //we need to see it in 2D instead of 1D
    //and we caculate where the current block point to
    int block_start_y = blockIdx.x / width_grid * TILE_WIDTH;
    int block_start_x = blockIdx.x % width_grid * TILE_WIDTH;

    //where this thread point to in 
    int row_idx = block_start_y + threadIdx.y;
    int col_idx = block_start_x + threadIdx.x;
    
    //this s_m size : (TILE_WIDTH + height_kernel) * (TILE_WIDTH + width_kernel) + height_kernel * width_kernel
    extern __shared__ float s_m[];
    float * temp_input = (float*)&s_m[0];
    float * temp_kernel = (float*)&s_m[(TILE_WIDTH + height_kernel) * (TILE_WIDTH + width_kernel)];

    //local 
    int r = threadIdx.y;
    int c = threadIdx.x;


    float accumulator =  bias_data[out_channel_ith];

    //loop each channel 
    int i,j;
    for (int in_channel_ith = 0; in_channel_ith < channel_in; in_channel_ith++){
        //read kernal for its channel 

        // for ( i = r ;i<height_kernel; i+= TILE_WIDTH){
        //     for ( j = c ; j < width_kernel; j+= TILE_WIDTH){
        //         temp_kernel[i*width_kernel + j] = weight_data[out_channel_ith*(channel_in*width_kernel*height_kernel) +
        //                                                     in_channel_ith*(width_kernel*height_kernel) + i*width_kernel + j];
        //     }
        // }

        //load data to shared mem 
        for ( i = r ;i<height_kernel+ TILE_WIDTH -1; i+= TILE_WIDTH){
            for ( j = c ; j < width_tiled ; j+= TILE_WIDTH){
                if(block_start_y  + i < height_in && block_start_x + j < width_in){
                    temp_input[i*width_tiled + j] = input_data[//batch_idx * (channel_in*width_in*height_in)
                                                            in_channel_ith*(width_in*height_in) + 
                                                            (block_start_y  + i)*width_in + block_start_x + j];
                }
            }
        }
        __syncthreads();
        //calculate 
        for ( i = 0 ;i<height_kernel; i++){
            for ( j = 0 ; j < width_kernel; j++){
                if (row_idx < height_out && col_idx < width_out) {
                    accumulator += temp_input[(i+r)*width_tiled + j+c] * weight_data[out_channel_ith*(channel_in*width_kernel*height_kernel) +
                                                                        in_channel_ith*(width_kernel*height_kernel) + i*width_kernel + j]; //temp_kernel[i*width_kernel + j];
                }
            }
        }
        __syncthreads();
    }
    __syncthreads();
    if (row_idx < height_out && col_idx < width_out)
    {
        output_data[//(batch_idx * (channel_out * height_out * width_out)) +
               (out_channel_ith * (height_out * width_out)) +
               (row_idx * width_out) +
               col_idx] = accumulator;
    }
}

                                     
__host__ void Kernel_simple_improved::cuda_conv_forward(int n_samples,  int channel_in,  int height_in, int width_in,
                                    int height_kernel, int width_kernel,  int channel_out,
                                     float *input_data,  float *weight_data,float *bias_data, float *output_data)
{
    const int height_out = height_in - height_kernel + 1;
    const int width_out = width_in - width_kernel + 1;

    // Allocate device memory
    float *device_input, *device_output, *device_weight,*device_bias;
    CHECK(hipMalloc((void **)&device_input, n_samples * channel_in * height_in * width_in * sizeof(float)));
    CHECK(hipMalloc((void **)&device_output, n_samples * channel_out * height_out * width_out * sizeof(float)));
    if (channel_out * channel_in * height_kernel * width_kernel < MAX_CONSTANT_SIZE){
        printf("Using constant!\n");
        CHECK(hipMemcpyToSymbol(HIP_SYMBOL(dc_weight), weight_data, channel_out * channel_in * height_kernel * width_kernel * sizeof(float)));
        device_weight = dc_weight;
        printf("pointer is %p %p\n",device_weight, dc_weight);
    }else{
        CHECK(hipMalloc((void **)&device_weight, channel_out * channel_in * height_kernel * width_kernel * sizeof(float)));
        CHECK(hipMemcpy(device_weight, weight_data, channel_out * channel_in * height_kernel * width_kernel * sizeof(float), hipMemcpyHostToDevice));
    }
    CHECK(hipMalloc((void **)&device_bias, channel_out * sizeof(float)));

    // Copy input and mask data to device
    CHECK(hipMemcpy(device_input, input_data, n_samples * channel_in * height_in * width_in * sizeof(float), hipMemcpyHostToDevice));
    
    CHECK(hipMemcpy(device_bias, bias_data, channel_out * sizeof(float), hipMemcpyHostToDevice));

    // Set the kernel dimensions and call the kernel
    int height_grid = (height_out - 1) / TILE_WIDTH + 1;
    int width_grid = (width_out - 1) / TILE_WIDTH + 1;
    int Z = height_grid * width_grid;
    dim3 num_threads_per_block(TILE_WIDTH, TILE_WIDTH, 1);
    dim3 num_blocks_in_grid(Z, channel_out,1);

    // Launch the kernel
    int share_mem_size = ((TILE_WIDTH + height_kernel) * (TILE_WIDTH + width_kernel) + height_kernel * width_kernel) * sizeof(float);
    for (int i = 0; i < n_samples; i ++) {
        conv_forward_kernel_2<<<num_blocks_in_grid, num_threads_per_block,share_mem_size>>>( channel_in, height_in,  width_in, height_kernel, 
                             width_kernel,  height_out,  width_out,  channel_out,
                            device_input + i*channel_in * height_in * width_in,  device_weight,device_bias, device_output + i*channel_out * height_out * width_out);
    }
    CHECK(hipDeviceSynchronize()); // Ensure that the GPU has completed the computation

    // Copy the output back to host
    CHECK(hipMemcpy(output_data, device_output, n_samples * channel_out * height_out * width_out * sizeof(float), hipMemcpyDeviceToHost));

    // Free device memory
    CHECK(hipFree(device_input));
    CHECK(hipFree(device_output));
    CHECK(hipFree(device_weight));
}
